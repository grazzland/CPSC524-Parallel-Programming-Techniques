#include "hip/hip_runtime.h"
#define FP float

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

__global__ void gpu_matrixmult(FP *a,FP *b, FP *c, int n, int m, int p) {

  int col = threadIdx.x + blockDim.x * blockIdx.x;
  int row = threadIdx.y + blockDim.y * blockIdx.y;

  int indexb = col;
  int index = row * m + col;
  
  if(col < m && row < n) {
    c[index] = 0.;
    for (int indexa = row*p; indexa < (row*p + p); indexa++, indexb+=m) 
      c[index] += a[indexa]*b[indexb];
  }

}


void cpu_matrixmult(FP *a,FP *b, FP *c, int n, int m, int p) {

  int index, indexa, indexb;
  FP cvalue;
  for(int col=0;col < m; col++)
    for(int row=0;row < n; row++) {
      indexb = col;
      index = row * m + col;
      cvalue = 0.;
      for (indexa = row*p; indexa < (row*p + p); indexa++, indexb+=m) 
  cvalue += a[indexa]*b[indexb];
      c[index] -= cvalue; //NOTE: This calculates the diff between CPU and GPU computations.
    }
}


int main(int argc, char *argv[]) {

  int i, j; // loop counters

  int gpucount = 0; // Count of available GPUs
  int gpunum = 0; // Device number to use
  int Block_Dim = 1; //Block dimension, x and y, square

  int n, m, p; // matrix dimension
  FP *a,*b,*c;
  FP *dev_a, *dev_b, *dev_c;
  int size; // number of bytes in arrays

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // --------------------SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  else {
     printf("Device count = %d\n",gpucount);
  }

  if (argc!=6) {
    printf("Usage: matmul <matrix dim n> <matrix dim m> <matrix dim p> <block dim>\n");
    exit (-1);
  }

  n = atoi(argv[1]);
  m = atoi(argv[2]);
  p = atoi(argv[3]);

  Block_Dim = atoi(argv[4]); // Square block
  if (Block_Dim*Block_Dim > 1024) {
    printf("Error, too many threads in block\n");
    exit (-1);
  }
  Grid_Dim_m = (m - 1) / Block_Dim + 1;
  Grid_Dim_n = (n - 1) / Block_Dim + 1;

  hipSetDevice(gpunum);
  printf("Using device %d\n",gpunum);
  
  printf("Matrix Dimension = %d\n",n);
  printf("Block_Dim = %d, Grid_Dim[m, n] = [%d, %d]\n",Block_Dim,Grid_Dim_m,Grid_Dim_n);

  dim3 Grid(Grid_Dim_m, Grid_Dim_n); //Grid structure
  dim3 Block(Block_Dim, Block_Dim); //Block structure

  a = (FP *) malloc(n * p * sizeof(FP)); // dynamically allocated memory for arrays on host
  b = (FP *) malloc(p * m * sizeof(FP));
  c = (FP *) malloc(n * m * sizeof(FP)); // results from GPU

  srand(12345);
  // int p = n; //Used here only to illustrate proper initialization for non-square case
  for(i=0;i < n;i++)
    for(j=0;j < p;j++) {
      a[i * p + j] = (FP) rand() / (FP) RAND_MAX;
      //      a[i * p + j] = (FP) i+j; // may be helpful for debugging
    }

  for(i=0;i < p;i++)
    for(j=0;j < m;j++) {
      b[i * m + j] = (FP) rand() / (FP) RAND_MAX;
      //      b[i * n + j] = (FP) i+j; // may be helpful for debugging
    }

  // ------------- COMPUTATION DONE ON GPU ----------------------------

  hipMalloc((void**)&dev_a, size); // allocate memory on device
  hipMalloc((void**)&dev_b, size);
  hipMalloc((void**)&dev_c, size);

  hipMemcpy(dev_a, a , size ,hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b , size ,hipMemcpyHostToDevice);

  hipEventCreate(&start); // instrument code to measure start time
  hipEventCreate(&stop);
  
  hipEventRecord(start, 0);
  // hipEventSynchronize(start); // not needed

  gpu_matrixmult<<<Grid,Block>>>(dev_a,dev_b,dev_c,n);

  hipEventRecord(stop, 0); // instrument code to measure end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  hipMemcpy(c,dev_c, size ,hipMemcpyDeviceToHost);

  printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time

  // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
  // DEBUGGING USE ONLY (AND FOR LIMITED NUMBERS OF TIMING RUNS)

  hipEventRecord(start, 0); // use same timing
  // hipEventSynchronize(start); // not needed


  cpu_matrixmult(a,b,c, n); // do calculation on host (NOTE: This computes the diff with GPU result.)

  hipEventRecord(stop, 0); // instrument code to measue end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms); // exec. time

// ------------------- check device creates correct results -----------------

  double error, suma, sumb, sumc, ai, bi, ci;
  suma = 0.; sumb = 0; sumc = 0;
  for(i=0;i < n*n;i++) {
    ai = (double) a[i];
    bi = (double) b[i];
    ci = (double) c[i];
    suma += ai*ai;
    sumb += bi*bi;
    sumc += ci*ci;
  }
  suma = sqrt(suma);
  sumb = sqrt(sumb);
  sumc = sqrt(sumc);
  error =  sumc/(n*suma*sumb);
  printf("Scaled error between GPU and CPU: %e\n", error);

// -------------- clean up ---------------------------------------

  free(a);
  free(b);
  free(c);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
