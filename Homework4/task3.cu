#define FP float
#define TW 3
#define NTB 2

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  if (code != hipSuccess)  {
    fprintf(stderr,"GPU Error: %s %s %d\n", hipGetErrorString(code), file, line);
    // if (abort) exit(code);
  }
}


__global__ void gpu_matrixmult(FP *a, FP *b, FP *c, int n, int m, int p, int TW1, int NTB1) {

  extern __shared__ FP bigarray[];
  FP *atile = &bigarray[0], *btile = &bigarray[TW * TW], *cvalue = &bigarray[(NTB + 1) * TW * TW];
  int tx = threadIdx.x; int ty = threadIdx.y;
  int row = ty + TW * blockIdx.y;

  for (int kt = 0; kt < NTB; kt++) cvalue[kt * TW * TW + ty * TW + tx] = 0.;

  for (int k = 0; k < (TW + p - 1)/TW; k++) {
    atile[ty * TW + tx] = (k*TW + tx < p && row < n) ? a[row*p + k*TW + tx] : 0.;
    for (int kt = 0; kt < NTB; kt++) {
      int col = tx + TW * (blockIdx.x + kt);
      btile[ty * TW + tx] = (k*TW + ty < p && col < m) ? b[(k*TW + ty)*m + col] : 0.;
      __syncthreads();
      for (int i = 0; i < TW; ++i) cvalue[kt * TW * TW + ty * TW + tx] += atile[ty * TW + i] * btile[i * TW + tx];
      __syncthreads();
    }
  }

  for(int kt = 0; kt < NTB; kt++) {
    int col = tx + TW * (blockIdx.x + kt);
    if (row < n && col < m) c[row * m + col] = cvalue[kt * TW * TW + ty * TW + tx];
  }

}


void cpu_matrixmult(FP *a,FP *b, FP *c, int n, int m, int p) {

  size_t index, indexa, indexb;
  FP cvalue;
  for(int col=0;col < m; col++)
    for(int row=0;row < n; row++) {
      indexb = col;
      index = row * m + col;
      cvalue = 0.;
      for (indexa = row*p; indexa < (row*p + p); indexa++, indexb+=m) 
        cvalue += a[indexa]*b[indexb];
      c[index] -= cvalue; //NOTE: This calculates the diff between CPU and GPU computations.
    }
}


int main(int argc, char *argv[]) {

  int i, j; // loop counters

  int gpucount = 0; // Count of available GPUs
  int gpunum = 0; // Device number to use
  int Grid_Dim_m = 1, Grid_Dim_n = 1; //Grid dimension, x and y, square
  int Block_Dim = 1; //Block dimension, x and y, square

  int n, m, p; // matrix dimension
  FP *a,*b,*c;
  FP *dev_a, *dev_b, *dev_c;
  size_t size_a, size_b, size_c;

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // --------------------SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  else {
     printf("Device count = %d\n",gpucount);
  }

  if (argc!=6) {
    printf("Usage: matmul <matrix dim: n> <matrix dim: m> <matrix dim: p> <block dim> <adjacent tiles>\n");
    exit (-1);
  }

  n = atoi(argv[1]);
  m = atoi(argv[2]);
  p = atoi(argv[3]);
  //int NTB = atoi(argv[5]);

  Block_Dim = atoi(argv[4]); // Square block
  // int TW = Block_Dim;
  if (Block_Dim*Block_Dim > 1024) {
    printf("Error, too many threads in block\n");
    exit (-1);
  }

  Grid_Dim_m = (m - 1) / Block_Dim + 1;
  Grid_Dim_n = (n - 1) / Block_Dim + 1;

  hipSetDevice(gpunum);
  printf("Using device %d\n",gpunum);
  
  printf("Matrix Dimension = [%d, %d, %d]\n",n,m,p);
  printf("Block_Dim = %d, Grid_Dim[m, n] = [%d, %d]\n",Block_Dim,Grid_Dim_m,Grid_Dim_n);

  dim3 Grid(Grid_Dim_m, Grid_Dim_n); //Grid structure
  dim3 Block(Block_Dim, Block_Dim); //Block structure
  
  size_a = n * p * sizeof(FP);
  size_b = p * m * sizeof(FP);
  size_c = n * m * sizeof(FP);
  a = (FP *) malloc(size_a); // dynamically allocated memory for arrays on host
  b = (FP *) malloc(size_b);
  c = (FP *) malloc(size_c); // results from GPU
  printf("size_a = %zu\n", size_a);
  printf("size_b = %zu\n", size_b);
  printf("size_c = %zu\n", size_c);

  srand(12345);
  // int p = n; //Used here only to illustrate proper initialization for non-square case
  printf("a = \n");
  for(i=0;i < n;i++) {
    for(j=0;j < p;j++) {
      //a[i * p + j] = (FP) rand() / (FP) RAND_MAX;
      a[i * p + j] = (FP) i+j; // may be helpful for debugging
      printf("%.5f\t", a[i * p + j]); 
    }
     printf("\n");
  }

  printf("b = \n");
  for(i=0;i < p;i++) {
    for(j=0;j < m;j++) {
      //b[i * m + j] = (FP) rand() / (FP) RAND_MAX;
       b[i * m + j] = (FP) i+j; // may be helpful for debugging
       printf("%.5f\t", b[i * m + j]);
    }
     printf("\n");
  }
  
  for(i=0;i < n;i++)
    for(j=0;j < m;j++)
      c[i * m + j] = 0.;

  // ------------- COMPUTATION DONE ON GPU ----------------------------

  gpuErrchk(hipMalloc((void**)&dev_a, size_a)); // allocate memory on device
  gpuErrchk(hipMalloc((void**)&dev_b, size_b));
  gpuErrchk(hipMalloc((void**)&dev_c, size_c));

  gpuErrchk(hipMemcpy(dev_a, a , size_a ,hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(dev_b, b , size_b ,hipMemcpyHostToDevice));

  hipEventCreate(&start); // instrument code to measure start time
  hipEventCreate(&stop);
  
  hipEventRecord(start, 0);
  // cudaEventSynchronize(start); // not needed

  size_t Ns = ((2 * NTB + 1) * TW * TW) * sizeof(FP);
  gpu_matrixmult<<<Grid,Block,Ns>>>(dev_a,dev_b,dev_c,n,m,p,TW, NTB);

  hipEventRecord(stop, 0); // instrument code to measure end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  gpuErrchk(hipMemcpy(c,dev_c, size_c ,hipMemcpyDeviceToHost));

  printf("c0 = \n");
  for(i=0;i < n;i++) {
    for(j=0;j < m;j++) {
        printf("%.5f\t", c[i * m + j]);
    }
    printf("\n");
  }

  printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time

  // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
  // DEBUGGING USE ONLY (AND FOR LIMITED NUMBERS OF TIMING RUNS)

  hipEventRecord(start, 0); // use same timing
  // cudaEventSynchronize(start); // not needed


  cpu_matrixmult(a,b,c, n,m,p); // do calculation on host (NOTE: This computes the diff with GPU result.)

  printf("c1 = \n");
  for(i=0;i < n;i++) {
    for(j=0;j < m;j++) {
        printf("%.5f\t", c[i * m + j]);
    }
    printf("\n");
  }

  hipEventRecord(stop, 0); // instrument code to measue end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms); // exec. time

// ------------------- check device creates correct results -----------------
  double error, suma, sumb, sumc, ai, bi, ci;
  suma = 0.; sumb = 0; sumc = 0;
  for(size_t i=0;i<n*p;i++) {
    ai = (double) a[i];
    suma += ai * ai;
  }
  for(size_t i=0;i<p*m;i++) {
    bi = (double) b[i];
    sumb += bi * bi;
  }
  for(size_t i=0;i<n*m;i++) {
    ci = (double) c[i];
    sumc += ci*ci;
  }
  suma = sqrt(suma);
  sumb = sqrt(sumb);
  sumc = sqrt(sumc);
  error = sumc/(sqrt(n*m)*suma*sumb);
  printf("Scaled error between GPU and CPU: %e\n", error);

// -------------- clean up ---------------------------------------

  free(a);
  free(b);
  free(c);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
