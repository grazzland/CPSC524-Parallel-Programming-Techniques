#define FP float
#define TW 32

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

__global__ void gpu_matrixmult(FP *a,FP *b, FP *c, int n, int m, int p) {

  __shared__ FP atile[TW][TW], btile[TW][TW];
  int tx = threadIdx.x; int ty = threadIdx.y; FP cvalue = 0;
  int col = tx + blockDim.x * blockIdx.x;
  int row = ty + blockDim.y * blockIdx.y;
  
  if(col < m && row < n) {
    for (int i = 0; i <= (p - 1) / TW; i++) {
      atile[ty][tx] = a[row * p + i * TW + tx];
      btile[ty][tx] = b[(i * TW + ty) * m + col];
      __syncthreads();
      for (int k = 0; k < TW; k++) cvalue += atile[ty][k] * btile[k][tx];
      __syncthreads();
    }
    c[row * m + col] = cvalue;
  }
}


void cpu_matrixmult(FP *a,FP *b, FP *c, int n, int m, int p) {

  size_t index, indexa, indexb;
  FP cvalue;
  for(int col=0;col < m; col++)
    for(int row=0;row < n; row++) {
      indexb = col;
      index = row * m + col;
      cvalue = 0.;
      for (indexa = row*p; indexa < (row*p + p); indexa++, indexb+=m) 
        cvalue += a[indexa]*b[indexb];
      c[index] -= cvalue; //NOTE: This calculates the diff between CPU and GPU computations.
    }
}


int main(int argc, char *argv[]) {

  int i, j; // loop counters

  int gpucount = 0; // Count of available GPUs
  int gpunum = 0; // Device number to use
  int Grid_Dim_m = 1, Grid_Dim_n = 1; //Grid dimension, x and y, square
  int Block_Dim = 1; //Block dimension, x and y, square

  int n, m, p; // matrix dimension
  FP *a,*b,*c;
  FP *dev_a, *dev_b, *dev_c;
  size_t size_a, size_b, size_c;

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // --------------------SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  else {
     printf("Device count = %d\n",gpucount);
  }

  if (argc!=5) {
    printf("Usage: matmul <matrix dim n> <matrix dim m> <matrix dim p> <block dim>\n");
    exit (-1);
  }

  n = atoi(argv[1]);
  m = atoi(argv[2]);
  p = atoi(argv[3]);

  Block_Dim = atoi(argv[4]); // Square block
  if (Block_Dim*Block_Dim > 1024) {
    printf("Error, too many threads in block\n");
    exit (-1);
  }

  Grid_Dim_m = (m - 1) / Block_Dim + 1;
  Grid_Dim_n = (n - 1) / Block_Dim + 1;

  hipSetDevice(gpunum);
  printf("Using device %d\n",gpunum);
  
  printf("Matrix Dimension = [%d, %d, %d]\n",n,m,p);
  printf("Block_Dim = %d, Grid_Dim[m, n] = [%d, %d]\n",Block_Dim,Grid_Dim_m,Grid_Dim_n);

  dim3 Grid(Grid_Dim_m, Grid_Dim_n); //Grid structure
  dim3 Block(Block_Dim, Block_Dim); //Block structure
  
  size_a = n * p * sizeof(FP);
  size_b = p * m * sizeof(FP);
  size_c = n * m * sizeof(FP);
  a = (FP *) malloc(size_a); // dynamically allocated memory for arrays on host
  b = (FP *) malloc(size_b);
  c = (FP *) malloc(size_c); // results from GPU
  printf("size_a = %zu\n", size_a);
  printf("size_b = %zu\n", size_b);
  printf("size_c = %zu\n", size_c);

  srand(12345);
  // int p = n; //Used here only to illustrate proper initialization for non-square case
  for(i=0;i < n;i++)
    for(j=0;j < p;j++) {
      a[i * p + j] = (FP) rand() / (FP) RAND_MAX;
      //      a[i * p + j] = (FP) i+j; // may be helpful for debugging
    }

  for(i=0;i < p;i++)
    for(j=0;j < m;j++) {
      b[i * m + j] = (FP) rand() / (FP) RAND_MAX;
      //      b[i * n + j] = (FP) i+j; // may be helpful for debugging
    }

  // ------------- COMPUTATION DONE ON GPU ----------------------------

  hipMalloc((void**)&dev_a, size_a); // allocate memory on device
  hipMalloc((void**)&dev_b, size_b);
  hipMalloc((void**)&dev_c, size_c);

  hipMemcpy(dev_a, a , size_a ,hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b , size_b ,hipMemcpyHostToDevice);

  hipEventCreate(&start); // instrument code to measure start time
  hipEventCreate(&stop);
  
  hipEventRecord(start, 0);
  // cudaEventSynchronize(start); // not needed

  gpu_matrixmult<<<Grid,Block>>>(dev_a,dev_b,dev_c,n,m,p);

  hipEventRecord(stop, 0); // instrument code to measure end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  hipMemcpy(c,dev_c, size_c ,hipMemcpyDeviceToHost);

  printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time

  // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
  // DEBUGGING USE ONLY (AND FOR LIMITED NUMBERS OF TIMING RUNS)

  hipEventRecord(start, 0); // use same timing
  // cudaEventSynchronize(start); // not needed


  //cpu_matrixmult(a,b,c, n,m,p); // do calculation on host (NOTE: This computes the diff with GPU result.)

  hipEventRecord(stop, 0); // instrument code to measue end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms); // exec. time

// ------------------- check device creates correct results -----------------
  double error, suma, sumb, sumc, ai, bi, ci;
  suma = 0.; sumb = 0; sumc = 0;
  for(size_t i=0;i<n*p;i++) {
    ai = (double) a[i];
    suma += ai * ai;
  }
  for(size_t i=0;i<p*m;i++) {
    bi = (double) b[i];
    sumb += bi * bi;
  }
  for(size_t i=0;i<n*m;i++) {
    ci = (double) c[i];
    sumc += ci*ci;
  }
  suma = sqrt(suma);
  sumb = sqrt(sumb);
  sumc = sqrt(sumc);
  error = sumc/(sqrt(n*m)*suma*sumb);
  printf("Scaled error between GPU and CPU: %e\n", error);

// -------------- clean up ---------------------------------------

  free(a);
  free(b);
  free(c);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
